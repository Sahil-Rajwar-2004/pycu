#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>


// CUDA kernels starts from here


// kernels for 16bit integers

__global__ void addKernel(const int16_t *x,const int16_t *y,int16_t *z){ *z = *x + *y; }
__global__ void addKernel(const int16_t *x,const int32_t *y,int32_t *z){ *z = *x + *y; }
__global__ void addKernel(const int16_t *x,const int64_t *y,int64_t *z){ *z = *x + *y; }
__global__ void subKernel(const int16_t *x,const int16_t *y,int16_t *z){ *z = *x - *y; }
__global__ void subKernel(const int16_t *x,const int32_t *y,int32_t *z){ *z = *x - *y; }
__global__ void subKernel(const int16_t *x,const int64_t *y,int64_t *z){ *z = *x - *y; }
__global__ void mulKernel(const int16_t *x,const int16_t *y,int16_t *z){ *z = *x * *y; }
__global__ void mulKernel(const int16_t *x,const int32_t *y,int32_t *z){ *z = *x * *y; }
__global__ void mulKernel(const int16_t *x,const int64_t *y,int64_t *z){ *z = *x * *y; }
__global__ void tdivKernel(const int16_t *x,const int16_t *y,int16_t *z){ *z = *x / *y; }
__global__ void tdivKernel(const int16_t *x,const int32_t *y,int32_t *z){ *z = *x / *y; }
__global__ void tdivKernel(const int16_t *x,const int64_t *y,int64_t *z){ *z = *x / *y; }
__global__ void fdivKernel(const int16_t *x,const int16_t *y,int16_t *z){ *z = *x / *y; }
__global__ void fdivKernel(const int16_t *x,const int32_t *y,int32_t *z){ *z = *x / *y; }
__global__ void fdivKernel(const int16_t *x,const int64_t *y,int64_t *z){ *z = *x / *y; }
__global__ void modKernel(const int16_t *x,const int16_t *y,int16_t *z){ *z = *x % *y; }
__global__ void modKernel(const int16_t *x,const int32_t *y,int32_t *z){ *z = *x % *y; }
__global__ void modKernel(const int16_t *x,const int64_t *y,int64_t *z){ *z = *x % *y; }
__global__ void powKernel(const int16_t *x,const int16_t *y,int16_t *z){ *z = pow(*x,*y); }
__global__ void powKernel(const int16_t *x,const int32_t *y,int32_t *z){ *z = pow(*x,*y); }
__global__ void powKernel(const int16_t *x,const int64_t *y,int64_t *z){ *z = pow(*x,*y); }
__global__ void absKernel(const int16_t *x,int16_t *z){
  if(*x < 0){ *z = -*x; }
  else{ *z = *x; }
}
__global__ void negKernel(const int16_t *x,int16_t *z){ *z = -*x; }
__global__ void posKernel(const int16_t *x,int16_t *z){ *z = +*x; }

__global__ void eqKernel(const int16_t *x,const int16_t *y,int16_t *z){ *z = (*x == *y) ? 1 : 0; }
__global__ void eqKernel(const int16_t *x,const int32_t *y,int32_t *z){ *z = (*x == *y) ? 1 : 0; }
__global__ void eqKernel(const int16_t *x,const int64_t *y,int64_t *z){ *z = (*x == *y) ? 1 : 0; }
__global__ void neKernel(const int16_t *x,const int16_t *y,int16_t *z){ *z = (*x != *y) ? 1 : 0; }
__global__ void neKernel(const int16_t *x,const int32_t *y,int32_t *z){ *z = (*x != *y) ? 1 : 0; }
__global__ void neKernel(const int16_t *x,const int64_t *y,int64_t *z){ *z = (*x != *y) ? 1 : 0; }
__global__ void gtKernel(const int16_t *x,const int16_t *y,int16_t *z){ *z = (*x > *y) ? 1 : 0; }
__global__ void gtKernel(const int16_t *x,const int32_t *y,int32_t *z){ *z = (*x > *y) ? 1 : 0; }
__global__ void gtKernel(const int16_t *x,const int64_t *y,int64_t *z){ *z = (*x > *y) ? 1 : 0; }
__global__ void geKernel(const int16_t *x,const int16_t *y,int16_t *z){ *z = (*x > *y) ? 1 : 0; }
__global__ void geKernel(const int16_t *x,const int32_t *y,int32_t *z){ *z = (*x >= *y) ? 1 : 0; }
__global__ void geKernel(const int16_t *x,const int64_t *y,int64_t *z){ *z = (*x >= *y) ? 1 : 0; }
__global__ void ltKernel(const int16_t *x,const int16_t *y,int16_t *z){ *z = (*x < *y) ? 1 : 0; }
__global__ void ltKernel(const int16_t *x,const int32_t *y,int32_t *z){ *z = (*x < *y) ? 1 : 0; }
__global__ void ltKernel(const int16_t *x,const int64_t *y,int64_t *z){ *z = (*x < *y) ? 1 : 0; }
__global__ void leKernel(const int16_t *x,const int16_t *y,int16_t *z){ *z = (*x <= *y) ? 1 : 0; }
__global__ void leKernel(const int16_t *x,const int32_t *y,int32_t *z){ *z = (*x <= *y) ? 1 : 0; }
__global__ void leKernel(const int16_t *x,const int64_t *y,int64_t *z){ *z = (*x <= *y) ? 1 : 0; }


// kerneles for 32bit integers 

__global__ void addKernel(const int32_t *x,const int32_t *y,int32_t *z){ *z = *x + *y; }
__global__ void addKernel(const int32_t *x,const int16_t *y,int32_t *z){ *z = *x + *y; }
__global__ void addKernel(const int32_t *x,const int64_t *y,int64_t *z){ *z = *x + *y; }
__global__ void subKernel(const int32_t *x,const int32_t *y,int32_t *z){ *z = *x - *y; }
__global__ void subKernel(const int32_t *x,const int16_t *y,int32_t *z){ *z = *x - *y; } 
__global__ void subKernel(const int32_t *x,const int64_t *y,int64_t *z){ *z = *x - *y; }
__global__ void mulKernel(const int32_t *x,const int32_t *y,int32_t *z){ *z = *x * *y; }
__global__ void mulKernel(const int32_t *x,const int16_t *y,int32_t *z){ *z = *x * *y; }
__global__ void mulKernel(const int32_t *x,const int64_t *y,int64_t *z){ *z = *x * *y; }
__global__ void tdivKernel(const int32_t *x,const int32_t *y,int32_t *z){ *z = *x / *y; }
__global__ void tdivKernel(const int32_t *x,const int16_t *y,int32_t *z){ *z = *x / *y; }
__global__ void tdivKernel(const int32_t *x,const int64_t *y,int64_t *z){ *z = *x / *y; }
__global__ void fdivKernel(const int32_t *x,const int32_t *y,int32_t *z){ *z = *x / *y; }
__global__ void fdivKernel(const int32_t *x,const int16_t *y,int32_t *z){ *z = *x / *y; }
__global__ void fdivKernel(const int32_t *x,const int64_t *y,int64_t *z){ *z = *x / *y; }
__global__ void modKernel(const int32_t *x,const int32_t *y,int32_t *z){ *z = *x % *y; }
__global__ void modKernel(const int32_t *x,const int16_t *y,int32_t *z){ *z = *x % *y; }
__global__ void modKernel(const int32_t *x,const int64_t *y,int64_t *z){ *z = *x % *y; }
__global__ void powKernel(const int32_t *x,const int32_t *y,int32_t *z){ *z = pow(*x,*y); }
__global__ void powKernel(const int32_t *x,const int16_t *y,int32_t *z){ *z = pow(*x,*y); }
__global__ void powKernel(const int32_t *x,const int64_t *y,int64_t *z){ *z = pow(*x,*y); }
__global__ void absKernel(const int32_t *x,int32_t *z){
  if(*x < 0){ *z = -*x; }
  else{ *z = *x; }
}
__global__ void negKernel(const int32_t *x,int32_t *z){ *z = -*x; }
__global__ void posKernel(const int32_t *x,int32_t *z){ *z = +*x; }

__global__ void eqKernel(const int32_t *x,const int32_t *y,int32_t *z){ *z = (*x == *y) ? 1 : 0; }
__global__ void eqKernel(const int32_t *x,const int16_t *y,int32_t *z){ *z = (*x == *y) ? 1 : 0; }
__global__ void eqKernel(const int32_t *x,const int64_t *y,int64_t *z){ *z = (*x == *y) ? 1 : 0; }
__global__ void neKernel(const int32_t *x,const int32_t *y,int32_t *z){ *z = (*x != *y) ? 1 : 0; }
__global__ void neKernel(const int32_t *x,const int16_t *y,int32_t *z){ *z = (*x != *y) ? 1 : 0; }
__global__ void neKernel(const int32_t *x,const int64_t *y,int64_t *z){ *z = (*x != *y) ? 1 : 0; }
__global__ void gtKernel(const int32_t *x,const int32_t *y,int32_t *z){ *z = (*x > *y) ? 1 : 0; }
__global__ void gtKernel(const int32_t *x,const int16_t *y,int32_t *z){ *z = (*x > *y) ? 1 : 0; }
__global__ void gtKernel(const int32_t *x,const int64_t *y,int64_t *z){ *z = (*x > *y) ? 1 : 0; }
__global__ void geKernel(const int32_t *x,const int32_t *y,int32_t *z){ *z = (*x >= *y) ? 1 : 0; }
__global__ void geKernel(const int32_t *x,const int16_t *y,int32_t *z){ *z = (*x >= *y) ? 1 : 0; }
__global__ void geKernel(const int32_t *x,const int64_t *y,int64_t *z){ *z = (*x >= *y) ? 1 : 0; }
__global__ void ltKernel(const int32_t *x,const int32_t *y,int32_t *z){ *z = (*x < *y) ? 1 : 0; }
__global__ void ltKernel(const int32_t *x,const int16_t *y,int32_t *z){ *z = (*x < *y) ? 1 : 0; }
__global__ void ltKernel(const int32_t *x,const int64_t *y,int64_t *z){ *z = (*x < *y) ? 1 : 0; }
__global__ void leKernel(const int32_t *x,const int32_t *y,int32_t *z){ *z = (*x <= *y) ? 1 : 0; }
__global__ void leKernel(const int32_t *x,const int16_t *y,int32_t *z){ *z = (*x <= *y) ? 1 : 0; }
__global__ void leKernel(const int32_t *x,const int64_t *y,int64_t *z){ *z = (*x <= *y) ? 1 : 0; }


// kernels for 64bit integers

__global__ void addKernel(const int64_t *x,const int16_t *y,int64_t *z){ *z = *x + *y; }
__global__ void addKernel(const int64_t *x,const int32_t *y,int64_t *z){ *z = *x + *y; }
__global__ void addKernel(const int64_t *x,const int64_t *y,int64_t *z){ *z = *x + *y; }
__global__ void subKernel(const int64_t *x,const int16_t *y,int64_t *z){ *z = *x - *y; }
__global__ void subKernel(const int64_t *x,const int32_t *y,int64_t *z){ *z = *x - *y; }
__global__ void subKernel(const int64_t *x,const int64_t *y,int64_t *z){ *z = *x - *y; }
__global__ void mulKernel(const int64_t *x,const int16_t *y,int64_t *z){ *z = *x * *y; }
__global__ void mulKernel(const int64_t *x,const int32_t *y,int64_t *z){ *z = *x * *y; }
__global__ void mulKernel(const int64_t *x,const int64_t *y,int64_t *z){ *z = *x * *y; }
__global__ void tdivKernel(const int64_t *x,const int16_t *y,int64_t *z){ *z = *x / *y; }
__global__ void tdivKernel(const int64_t *x,const int32_t *y,int64_t *z){ *z = *x / *y; }
__global__ void tdivKernel(const int64_t *x,const int64_t *y,int64_t *z){ *z = *x / *y; }
__global__ void fdivKernel(const int64_t *x,const int16_t *y,int64_t *z){ *z = *x / *y; }
__global__ void fdivKernel(const int64_t *x,const int32_t *y,int64_t *z){ *z = *x / *y; }
__global__ void fdivKernel(const int64_t *x,const int64_t *y,int64_t *z){ *z = *x / *y; }
__global__ void modKernel(const int64_t *x,const int16_t *y,int64_t *z){ *z = *x % *y; }
__global__ void modKernel(const int64_t *x,const int32_t *y,int64_t *z){ *z = *x % *y; }
__global__ void modKernel(const int64_t *x,const int64_t *y,int64_t *z){ *z = *x % *y; }
__global__ void powKernel(const int64_t *x,const int16_t *y,int64_t *z){ *z = pow(*x,*y); }
__global__ void powKernel(const int64_t *x,const int32_t *y,int64_t *z){ *z = pow(*x,*y); }
__global__ void powKernel(const int64_t *x,const int64_t *y,int64_t *z){ *z = pow(*x,*y); }
__global__ void absKernel(const int64_t *x,int64_t *z){
  if(*x < 0){ *z = -*x; }
  else{ *z = *x; }
}
__global__ void negKernel(const int64_t *x,int64_t *z){ *z = -*x; }
__global__ void posKernel(const int64_t *x,int64_t *z){ *z = +*x; }

__global__ void eqKernel(const int64_t *x,const int16_t *y,int64_t *z){ *z = (*x == *y) ? 1 : 0; }
__global__ void eqKernel(const int64_t *x,const int32_t *y,int64_t *z){ *z = (*x == *y) ? 1 : 0; }
__global__ void eqKernel(const int64_t *x,const int64_t *y,int64_t *z){ *z = (*x == *y) ? 1 : 0; }
__global__ void neKernel(const int64_t *x,const int16_t *y,int64_t *z){ *z = (*x != *y) ? 1 : 0; }
__global__ void neKernel(const int64_t *x,const int32_t *y,int64_t *z){ *z = (*x != *y) ? 1 : 0; }
__global__ void neKernel(const int64_t *x,const int64_t *y,int64_t *z){ *z = (*x != *y) ? 1 : 0; }
__global__ void gtKernel(const int64_t *x,const int16_t *y,int64_t *z){ *z = (*x > *y) ? 1 : 0; }
__global__ void gtKernel(const int64_t *x,const int32_t *y,int64_t *z){ *z = (*x > *y) ? 1 : 0; }
__global__ void gtKernel(const int64_t *x,const int64_t *y,int64_t *z){ *z = (*x > *y) ? 1 : 0; }
__global__ void geKernel(const int64_t *x,const int16_t *y,int64_t *z){ *z = (*x >= *y) ? 1 : 0; }
__global__ void geKernel(const int64_t *x,const int32_t *y,int64_t *z){ *z = (*x >= *y) ? 1 : 0; }
__global__ void geKernel(const int64_t *x,const int64_t *y,int64_t *z){ *z = (*x >= *y) ? 1 : 0; }
__global__ void ltKernel(const int64_t *x,const int16_t *y,int64_t *z){ *z = (*x < *y) ? 1 : 0; }
__global__ void ltKernel(const int64_t *x,const int32_t *y,int64_t *z){ *z = (*x < *y) ? 1 : 0; }
__global__ void ltKernel(const int64_t *x,const int64_t *y,int64_t *z){ *z = (*x < *y) ? 1 : 0; }
__global__ void leKernel(const int64_t *x,const int16_t *y,int64_t *z){ *z = (*x <= *y) ? 1 : 0; }
__global__ void leKernel(const int64_t *x,const int32_t *y,int64_t *z){ *z = (*x <= *y) ? 1 : 0; }
__global__ void leKernel(const int64_t *x,const int64_t *y,int64_t *z){ *z = (*x <= *y) ? 1 : 0; }


// CUDA kernels ends here




class int16 ;
class int32 ;
class int64 ;


class int16{
  private:
    int16_t *value;

  public:
    int16(const int16_t &other){
      hipMalloc(&value,sizeof(int16_t));
      hipMemcpy(value,&other,sizeof(int16_t),hipMemcpyHostToDevice);
    };

    ~int16(){
      if(value){ hipFree(value); }
    };

    int16_t getValue() const {
      int16_t host_value;
      hipMemcpy(&host_value,value,sizeof(int16_t),hipMemcpyDeviceToHost);
      return host_value;
    }

    int16_t *deviceValue() const { return this -> value; }

    int16_t *copyToHost() const {
      int16_t *host_value = new int16_t;
      hipMemcpy(host_value,value,sizeof(int16_t),hipMemcpyDeviceToHost);
      return host_value;
    }

    int16 add(const int16 &other) const ;
    int16 sub(const int16 &other) const ;
    int16 mul(const int16 &other) const ;
    int16 tdiv(const int16 &other) const ;
    int16 fdiv(const int16 &other) const ;
    int16 mod(const int16 &other) const ;
    int16 pow(const int16 &other) const;

    int16_t eq(const int16 &other) const ;
    int16_t ne(const int16 &other) const ;
    int16_t gt(const int16 &other) const ;
    int16_t ge(const int16 &other) const ;
    int16_t lt(const int16 &other) const ;
    int16_t le(const int16 &other) const ;
    
    int16 abs() const ;
    int16 neg() const ;
    int16 pos() const ;

    int32 add(const int32 &other) const ;
    int32 sub(const int32 &other) const ;
    int32 mul(const int32 &other) const ;
    int32 tdiv(const int32 &other) const ;
    int32 fdiv(const int32 &other) const ;
    int32 mod(const int32 &other) const ;
    int32 pow(const int32 &other) const ;

    int32_t eq(const int32 &other) const ;
    int32_t ne(const int32 &other) const ;
    int32_t gt(const int32 &other) const ;
    int32_t ge(const int32 &other) const ;
    int32_t lt(const int32 &other) const ;
    int32_t le(const int32 &other) const ;

    int64 add(const int64 &other) const ;
    int64 sub(const int64 &other) const ;
    int64 mul(const int64 &other) const ;
    int64 tdiv(const int64 &other) const ;
    int64 fdiv(const int64 &other) const ;
    int64 mod(const int64 &other) const ;
    int64 pow(const int64 &other) const ;

    int64_t eq(const int64 &other) const ;
    int64_t ne(const int64 &other) const ;
    int64_t gt(const int64 &other) const ;
    int64_t ge(const int64 &other) const ;
    int64_t lt(const int64 &other) const ;
    int64_t le(const int64 &other) const ;
};


class int32{
  private:
  int32_t *value;

  public:
    int32(const int32_t &other){
      hipMalloc(&value,sizeof(int32_t));
      hipMemcpy(value,&other,sizeof(int32_t),hipMemcpyHostToDevice);
    };

    ~int32(){ if(value){ hipFree(value); } };

    int32_t getValue() const {
      int32_t host_value;
      hipMemcpy(&host_value,value,sizeof(int32_t),hipMemcpyDeviceToHost);
      return host_value;
    }

    int32_t *deviceValue() const { return this -> value; }

    int32_t *copyToHost() const {
      int32_t *host_value = new int32_t;
      hipMemcpy(host_value,value,sizeof(int32_t),hipMemcpyDeviceToHost);
      return host_value;
    }

    int32 add(const int32 &other) const ;
    int32 sub(const int32 &other) const ;
    int32 mul(const int32 &other) const ;
    int32 tdiv(const int32 &other) const ;
    int32 fdiv(const int32 &other) const ;
    int32 mod(const int32 &other) const ;
    int32 pow(const int32 &other) const ;

    int32_t eq(const int32 &other) const ;
    int32_t ne(const int32 &other) const ;
    int32_t gt(const int32 &other) const ;
    int32_t ge(const int32 &other) const ;
    int32_t lt(const int32 &other) const ;
    int32_t le(const int32 &other) const ;

    int32 abs() const ;
    int32 neg() const ;
    int32 pos() const ;

    int32 add(const int16 &other) const ;
    int32 sub(const int16 &other) const ;
    int32 mul(const int16 &other) const ;
    int32 tdiv(const int16 &other) const ;
    int32 fdiv(const int16 &other) const ;
    int32 mod(const int16 &other) const ;
    int32 pow(const int16 &other) const ;

    int32_t eq(const int16 &other) const ;
    int32_t ne(const int16 &other) const ;
    int32_t gt(const int16 &other) const ;
    int32_t ge(const int16 &other) const ;
    int32_t lt(const int16 &other) const ;
    int32_t le(const int16 &other) const ;

    int64 add(const int64 &other) const ;
    int64 sub(const int64 &other) const ;
    int64 mul(const int64 &other) const ;
    int64 tdiv(const int64 &other) const ;
    int64 fdiv(const int64 &other) const ;
    int64 mod(const int64 &other) const ;
    int64 pow(const int64 &other) const ;

    int64_t eq(const int64 &other) const ;
    int64_t ne(const int64 &other) const ;
    int64_t gt(const int64 &other) const ;
    int64_t ge(const int64 &other) const ;
    int64_t lt(const int64 &other) const ;
    int64_t le(const int64 &other) const ;
};


class int64{
  private:
    int64_t *value;
  public:
    int64(const int64_t &other){
      hipMalloc(&value,sizeof(int64_t));
      hipMemcpy(value,&other,sizeof(int64_t),hipMemcpyHostToDevice);
    }

    ~int64(){ if(value){ hipFree(value); } }

    int64_t getValue() const {
      int64_t host_value;
      hipMemcpy(&host_value,value,sizeof(int64_t),hipMemcpyDeviceToHost);
      return host_value;
    }

    int64_t *deviceValue() const { return this -> value; }

    int64_t *copyToHost() const {
      int64_t *host_value = new int64_t;
      hipMemcpy(host_value,value,sizeof(int64_t),hipMemcpyDeviceToHost);
      return host_value;
    }

    int64 add(const int64 &other) const ;
    int64 sub(const int64 &other) const ;
    int64 mul(const int64 &other) const ;
    int64 tdiv(const int64 &other) const ;
    int64 fdiv(const int64 &other) const ;
    int64 mod(const int64 &other) const ;
    int64 pow(const int64 &other) const ;

    int64_t eq(const int64 &other) const ;
    int64_t ne(const int64 &other) const ;
    int64_t gt(const int64 &other) const ;
    int64_t ge(const int64 &other) const ;
    int64_t lt(const int64 &other) const ;
    int64_t le(const int64 &other) const ;

    int64 abs() const ;
    int64 neg() const ;
    int64 pos() const ;

    int64 add(const int32 &other) const ;
    int64 sub(const int32 &other) const ;
    int64 mul(const int32 &other) const ;
    int64 tdiv(const int32 &other) const ;
    int64 fdiv(const int32 &other) const ;
    int64 mod(const int32 &other) const ;
    int64 pow(const int32 &other) const ;

    int64_t eq(const int32 &other) const ;
    int64_t ne(const int32 &other) const ;
    int64_t gt(const int32 &other) const ;
    int64_t ge(const int32 &other) const ;
    int64_t lt(const int32 &other) const ;
    int64_t le(const int32 &other) const ;

    int64 add(const int16 &other) const ;
    int64 sub(const int16 &other) const ;
    int64 mul(const int16 &other) const ;
    int64 tdiv(const int16 &other) const ;
    int64 fdiv(const int16 &other) const ;
    int64 mod(const int16 &other) const ;
    int64 pow(const int16 &other) const ;

    int64_t eq(const int16 &other) const ;
    int64_t ne(const int16 &other) const ;
    int64_t gt(const int16 &other) const ;
    int64_t ge(const int16 &other) const ;
    int64_t lt(const int16 &other) const ;
    int64_t le(const int16 &other) const ;
};




// for int16

int16 int16::add(const int16 &other) const {
  int16 result(0);
  addKernel<<<1,1>>>(this -> value,other.value,result.value);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int16 int16::sub(const int16 &other) const {
  int16 result(0);
  subKernel<<<1,1>>>(this -> value,other.value,result.value);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int16 int16::mul(const int16 &other) const {
  int16 result(0);
  mulKernel<<<1,1>>>(this -> value,other.value,result.value);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int16 int16::tdiv(const int16 &other) const {
  int16 result(0);
  tdivKernel<<<1,1>>>(this -> value,other.value,result.value);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int16 int16::fdiv(const int16 &other) const {
  int16 result(0);
  fdivKernel<<<1,1>>>(this -> value,other.value,result.value);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int16 int16::mod(const int16 &other) const {
  int16 result(0);
  modKernel<<<1,1>>>(this -> value,other.value,result.value);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int16 int16::pow(const int16 &other) const {
  int16 result(0);
  powKernel<<<1,1>>>(this -> value,other.value,result.value);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int16 int16::abs() const {
  int16 result(0);
  absKernel<<<1,1>>>(this -> value,result.value);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int16 int16::neg() const {
  int16 result(0);
  negKernel<<<1,1>>>(this -> value,result.value);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int16 int16::pos() const {
  int16 result(0);
  posKernel<<<1,1>>>(this -> value,result.value);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int16_t int16::eq(const int16 &other) const {
  int16_t host_value;
  int16_t *device_value;
  hipMalloc(&device_value,sizeof(int16_t));
  eqKernel<<<1,1>>>(this -> value,other.value,device_value);
  hipGetLastError();
  hipDeviceSynchronize();
  hipMemcpy(&host_value,device_value,sizeof(int16_t),hipMemcpyDeviceToHost);
  hipFree(device_value);
  return host_value;
}

int16_t int16::ne(const int16 &other) const {
  int16_t host_value;
  int16_t *device_value;
  hipMalloc(&device_value,sizeof(int16_t));
  neKernel<<<1,1>>>(this -> value,other.value,device_value);
  hipGetLastError();
  hipDeviceSynchronize();
  hipMemcpy(&host_value,device_value,sizeof(int16_t),hipMemcpyDeviceToHost);
  hipFree(device_value);
  return host_value;
}

int16_t int16::gt(const int16 &other) const {
  int16_t host_value;
  int16_t *device_value;
  hipMalloc(&device_value,sizeof(int16_t));
  gtKernel<<<1,1>>>(this -> value,other.value,device_value);
  hipGetLastError();
  hipDeviceSynchronize();
  hipMemcpy(&host_value,device_value,sizeof(int16_t),hipMemcpyDeviceToHost);
  hipFree(device_value);
  return host_value;
}

int16_t int16::ge(const int16 &other) const {
  int16_t host_value;
  int16_t *device_value;
  hipMalloc(&device_value,sizeof(int16_t));
  geKernel<<<1,1>>>(this -> value,other.value,device_value);
  hipGetLastError();
  hipDeviceSynchronize();
  hipMemcpy(&host_value,device_value,sizeof(int16_t),hipMemcpyDeviceToHost);
  hipFree(device_value);
  return host_value;
}

int16_t int16::lt(const int16 &other) const {
  int16_t host_value;
  int16_t *device_value;
  hipMalloc(&device_value,sizeof(int16_t));
  ltKernel<<<1,1>>>(this -> value,other.value,device_value);
  hipGetLastError();
  hipDeviceSynchronize();
  hipMemcpy(&host_value,device_value,sizeof(int16_t),hipMemcpyDeviceToHost);
  hipFree(device_value);
  return host_value;
}

int16_t int16::le(const int16 &other) const {
  int16_t host_value;
  int16_t *device_value;
  hipMalloc(&device_value,sizeof(int16_t));
  leKernel<<<1,1>>>(this -> value,other.value,device_value);
  hipGetLastError();
  hipDeviceSynchronize();
  hipMemcpy(&host_value,device_value,sizeof(int16_t),hipMemcpyDeviceToHost);
  hipFree(device_value);
  return host_value;
}

int32 int16::add(const int32 &other) const {
  int32 result(0);
  addKernel<<<1,1>>>(this -> value,other.deviceValue(),result.deviceValue());
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}


int32 int16::sub(const int32 &other) const {
  int32 result(0);
  subKernel<<<1,1>>>(this -> value,other.deviceValue(),result.deviceValue());
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int32 int16::mul(const int32 &other) const {
  int32 result(0);
  mulKernel<<<1,1>>>(this -> value,other.deviceValue(),result.deviceValue());
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int32 int16::tdiv(const int32 &other) const {
  int32 result(0);
  tdivKernel<<<1,1>>>(this -> value,other.deviceValue(),result.deviceValue());
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int32 int16::fdiv(const int32 &other) const {
  int32 result(0);
  fdivKernel<<<1,1>>>(this -> value,other.deviceValue(),result.deviceValue());
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int32 int16::mod(const int32 &other) const {
  int32 result(0);
  modKernel<<<1,1>>>(this -> value,other.deviceValue(),result.deviceValue());
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int32 int16::pow(const int32 &other) const {
  int32 result(0);
  powKernel<<<1,1>>>(this -> value,other.deviceValue(),result.deviceValue());
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int32_t int16::eq(const int32 &other) const {
  int32_t host_value;
  int32_t *device_value;
  hipMalloc(&device_value,sizeof(int32_t));
  eqKernel<<<1,1>>>(this -> value,other.deviceValue(),device_value);
  hipGetLastError();
  hipDeviceSynchronize();
  hipMemcpy(&host_value,device_value,sizeof(int32_t),hipMemcpyDeviceToHost);
  hipFree(device_value);
  return host_value;
}

int32_t int16::ne(const int32 &other) const {
  int32_t host_value;
  int32_t *device_value;
  hipMalloc(&device_value,sizeof(int32_t));
  neKernel<<<1,1>>>(this -> value,other.deviceValue(),device_value);
  hipGetLastError();
  hipDeviceSynchronize();
  hipMemcpy(&host_value,device_value,sizeof(int32_t),hipMemcpyDeviceToHost);
  hipFree(device_value);
  return host_value;
}

int32_t int16::gt(const int32 &other) const {
  int32_t host_value;
  int32_t *device_value;
  hipMalloc(&device_value,sizeof(int32_t));
  gtKernel<<<1,1>>>(this -> value,other.deviceValue(),device_value);
  hipGetLastError();
  hipDeviceSynchronize();
  hipMemcpy(&host_value,device_value,sizeof(int32_t),hipMemcpyDeviceToHost);
  hipFree(device_value);
  return host_value;
}

int32_t int16::ge(const int32 &other) const {
  int32_t host_value;
  int32_t *device_value;
  hipMalloc(&device_value,sizeof(int32_t));
  geKernel<<<1,1>>>(this -> value,other.deviceValue(),device_value);
  hipGetLastError();
  hipDeviceSynchronize();
  hipMemcpy(&host_value,device_value,sizeof(int32_t),hipMemcpyDeviceToHost);
  hipFree(device_value);
  return host_value;
}

int32_t int16::lt(const int32 &other) const {
  int32_t host_value;
  int32_t *device_value;
  hipMalloc(&device_value,sizeof(int32_t));
  ltKernel<<<1,1>>>(this -> value,other.deviceValue(),device_value);
  hipGetLastError();
  hipDeviceSynchronize();
  hipMemcpy(&host_value,device_value,sizeof(int32_t),hipMemcpyDeviceToHost);
  hipFree(device_value);
  return host_value;
}

int32_t int16::le(const int32 &other) const {
  int32_t host_value;
  int32_t *device_value;
  hipMalloc(&device_value,sizeof(int32_t));
  leKernel<<<1,1>>>(this -> value,other.deviceValue(),device_value);
  hipGetLastError();
  hipDeviceSynchronize();
  hipMemcpy(&host_value,device_value,sizeof(int32_t),hipMemcpyDeviceToHost);
  hipFree(device_value);
  return host_value;
}

int64 int16::add(const int64 &other) const {
  int64 result(0);
  addKernel<<<1,1>>>(this -> value,other.deviceValue(),result.deviceValue());
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int64 int16::sub(const int64 &other) const {
  int64 result(0);
  subKernel<<<1,1>>>(this -> value,other.deviceValue(),result.deviceValue());
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int64 int16::mul(const int64 &other) const {
  int64 result(0);
  mulKernel<<<1,1>>>(this -> value,other.deviceValue(),result.deviceValue());
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int64 int16::tdiv(const int64 &other) const {
  int64 result(0);
  tdivKernel<<<1,1>>>(this -> value,other.deviceValue(),result.deviceValue());
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int64 int16::fdiv(const int64 &other) const {
  int64 result(0);
  fdivKernel<<<1,1>>>(this -> value,other.deviceValue(),result.deviceValue());
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int64 int16::mod(const int64 &other) const {
  int64 result(0);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int64 int16::pow(const int64 &other) const {
  int64 result(0);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int64_t int16::eq(const int64 &other) const {
  int64_t host_value;
  int64_t *device_value;
  hipMalloc(&device_value,sizeof(int64_t));
  eqKernel<<<1,1>>>(this -> value,other.deviceValue(),device_value);
  hipGetLastError();
  hipDeviceSynchronize();
  hipMemcpy(&host_value,device_value,sizeof(int64_t),hipMemcpyDeviceToHost);
  hipFree(device_value);
  return host_value;
}

int64_t int16::ne(const int64 &other) const {
  int64_t host_value;
  int64_t *device_value;
  hipMalloc(&device_value,sizeof(int64_t));
  neKernel<<<1,1>>>(this -> value,other.deviceValue(),device_value);
  hipGetLastError();
  hipDeviceSynchronize();
  hipMemcpy(&host_value,device_value,sizeof(int64_t),hipMemcpyDeviceToHost);
  hipFree(device_value);
  return host_value;
}

int64_t int16::gt(const int64 &other) const {
  int64_t host_value;
  int64_t *device_value;
  hipMalloc(&device_value,sizeof(int64_t));
  gtKernel<<<1,1>>>(this -> value,other.deviceValue(),device_value);
  hipGetLastError();
  hipDeviceSynchronize();
  hipMemcpy(&host_value,device_value,sizeof(int64_t),hipMemcpyDeviceToHost);
  hipFree(device_value);
  return host_value;
}

int64_t int16::ge(const int64 &other) const {
  int64_t host_value;
  int64_t *device_value;
  hipMalloc(&device_value,sizeof(int64_t));
  geKernel<<<1,1>>>(this -> value,other.deviceValue(),device_value);
  hipGetLastError();
  hipDeviceSynchronize();
  hipMemcpy(&host_value,device_value,sizeof(int64_t),hipMemcpyDeviceToHost);
  hipFree(device_value);
  return host_value;
}

int64_t int16::lt(const int64 &other) const {
  int64_t host_value;
  int64_t *device_value;
  hipMalloc(&device_value,sizeof(int64_t));
  ltKernel<<<1,1>>>(this -> value,other.deviceValue(),device_value);
  hipGetLastError();
  hipDeviceSynchronize();
  hipMemcpy(&host_value,device_value,sizeof(int64_t),hipMemcpyDeviceToHost);
  hipFree(device_value);
  return host_value;
}

int64_t int16::le(const int64 &other) const {
  int64_t host_value;
  int64_t *device_value;
  hipMalloc(&device_value,sizeof(int64_t));
  leKernel<<<1,1>>>(this -> value,other.deviceValue(),device_value);
  hipGetLastError();
  hipDeviceSynchronize();
  hipMemcpy(&host_value,device_value,sizeof(int64_t),hipMemcpyDeviceToHost);
  return host_value;
}

// for int32

int32 int32::add(const int32 &other) const {
  int32 result(0);
  addKernel<<<1,1>>>(this -> value,other.value,result.value);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int32 int32::sub(const int32 &other) const {
  int32 result(0);
  subKernel<<<1,1>>>(this -> value,other.value,result.value);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int32 int32::mul(const int32 &other) const {
  int32 result(0);
  mulKernel<<<1,1>>>(this -> value,other.value,result.value);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int32 int32::tdiv(const int32 &other) const {
  int32 result(0);
  tdivKernel<<<1,1>>>(this -> value,other.value,result.value);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int32 int32::fdiv(const int32 &other) const {
  int32 result(0);
  fdivKernel<<<1,1>>>(this -> value,other.value,result.value);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int32 int32::mod(const int32 &other) const {
  int32 result(0);
  modKernel<<<1,1>>>(this -> value,other.value,result.value);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int32 int32::pow(const int32 &other) const {
  int32 result(0);
  powKernel<<<1,1>>>(this -> value,other.value,result.value);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int32_t int32::eq(const int32 &other) const {
  int32_t host_value;
  int32_t *device_value;
  hipMalloc(&device_value,sizeof(int32_t));
  eqKernel<<<1,1>>>(this -> value,other.value,device_value);
  hipGetLastError();
  hipDeviceSynchronize();
  hipMemcpy(&host_value,device_value,sizeof(int32_t),hipMemcpyDeviceToHost);
  hipFree(device_value);
  return host_value;
}

int32_t int32::ne(const int32 &other) const {
  int32_t host_value;
  int32_t *device_value;
  hipMalloc(&device_value,sizeof(int32_t));
  neKernel<<<1,1>>>(this -> value,other.value,device_value);
  hipGetLastError();
  hipDeviceSynchronize();
  hipMemcpy(&host_value,device_value,sizeof(int32_t),hipMemcpyDeviceToHost);
  hipFree(device_value);
  return host_value;
}

int32_t int32::gt(const int32 &other) const {
  int32_t host_value;
  int32_t *device_value;
  hipMalloc(&device_value,sizeof(int32_t));
  gtKernel<<<1,1>>>(this -> value,other.value,device_value);
  hipGetLastError();
  hipDeviceSynchronize();
  hipMemcpy(&host_value,device_value,sizeof(int32_t),hipMemcpyDeviceToHost);
  hipFree(device_value);
  return host_value;
}

int32_t int32::ge(const int32 &other) const {
  int32_t host_value;
  int32_t *device_value;
  hipMalloc(&device_value,sizeof(int32_t));
  geKernel<<<1,1>>>(this -> value,other.value,device_value);
  hipGetLastError();
  hipDeviceSynchronize();
  hipMemcpy(&host_value,device_value,sizeof(int32_t),hipMemcpyDeviceToHost);
  hipFree(device_value);
  return host_value;
}

int32_t int32::lt(const int32 &other) const {
  int32_t host_value;
  int32_t *device_value;
  hipMalloc(&device_value,sizeof(int32_t));
  ltKernel<<<1,1>>>(this -> value,other.value,device_value);
  hipGetLastError();
  hipDeviceSynchronize();
  hipMemcpy(&host_value,device_value,sizeof(int32_t),hipMemcpyDeviceToHost);
  hipFree(device_value);
  return host_value;
}

int32_t int32::le(const int32 &other) const {
  int32_t host_value;
  int32_t *device_value;
  hipMalloc(&device_value,sizeof(int32_t));
  hipGetLastError();
  hipDeviceSynchronize();
  leKernel<<<1,1>>>(this -> value,other.value,device_value);
  hipGetLastError();
  hipDeviceSynchronize();
  hipMemcpy(&host_value,device_value,sizeof(int32_t),hipMemcpyDeviceToHost);
  hipFree(device_value);
  return host_value;
}

int32 int32::abs() const {
  int32 result(0);
  absKernel<<<1,1>>>(this -> value,result.value);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int32 int32::neg() const {
  int32 result(0);
  negKernel<<<1,1>>>(this -> value,result.value);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int32 int32::pos() const {
  int32 result(0);
  posKernel<<<1,1>>>(this -> value,result.value);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int32 int32::add(const int16 &other) const {
  int32 result(0);
  addKernel<<<1,1>>>(this -> value,other.deviceValue(),result.value);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int32 int32::sub(const int16 &other) const {
  int32 result(0);
  subKernel<<<1,1>>>(this -> value,other.deviceValue(),result.value);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int32 int32::mul(const int16 &other) const {
  int32 result(0);
  mulKernel<<<1,1>>>(this -> value,other.deviceValue(),result.value);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int32 int32::tdiv(const int16 &other) const {
  int32 result(0);
  tdivKernel<<<1,1>>>(this -> value,other.deviceValue(),result.value);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int32 int32::fdiv(const int16 &other) const {
  int32 result(0);
  fdivKernel<<<1,1>>>(this -> value,other.deviceValue(),result.value);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int32 int32::mod(const int16 &other) const {
  int32 result(0);
  modKernel<<<1,1>>>(this -> value,other.deviceValue(),result.value);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int32 int32::pow(const int16 &other) const {
  int32 result(0);
  powKernel<<<1,1>>>(this -> value,other.deviceValue(),result.value);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int32_t int32::eq(const int16 &other) const {
  int32_t host_value;
  int32_t *device_value;
  hipMalloc(&device_value,sizeof(int32_t));
  eqKernel<<<1,1>>>(this -> value,other.deviceValue(),device_value);
  hipGetLastError();
  hipDeviceSynchronize();
  hipMemcpy(&host_value,device_value,sizeof(int32_t),hipMemcpyDeviceToHost);
  hipFree(device_value);
  return host_value;
}

int32_t int32::ne(const int16 &other) const {
  int32_t host_value;
  int32_t *device_value;
  hipMalloc(&device_value,sizeof(int32_t));
  neKernel<<<1,1>>>(this -> value,other.deviceValue(),device_value);
  hipGetLastError();
  hipDeviceSynchronize();
  hipMemcpy(&host_value,device_value,sizeof(int32_t),hipMemcpyDeviceToHost);
  hipFree(device_value);
  return host_value;
}

int32_t int32::gt(const int16 &other) const {
  int32_t host_value;
  int32_t *device_value;
  hipMalloc(&device_value,sizeof(int32_t));
  gtKernel<<<1,1>>>(this -> value,other.deviceValue(),device_value);
  hipGetLastError();
  hipDeviceSynchronize();
  hipMemcpy(&host_value,device_value,sizeof(int32_t),hipMemcpyDeviceToHost);
  hipFree(device_value);
  return host_value;
}

int32_t int32::ge(const int16 &other) const {
  int32_t host_value;
  int32_t *device_value;
  hipMalloc(&device_value,sizeof(int32_t));
  geKernel<<<1,1>>>(this -> value,other.deviceValue(),device_value);
  hipGetLastError();
  hipDeviceSynchronize();
  hipMemcpy(&host_value,device_value,sizeof(int32_t),hipMemcpyDeviceToHost);
  hipFree(device_value);
  return host_value;
}

int32_t int32::lt(const int16 &other) const {
  int32_t host_value;
  int32_t *device_value;
  hipMalloc(&device_value,sizeof(int32_t));
  ltKernel<<<1,1>>>(this -> value,other.deviceValue(),device_value);
  hipGetLastError();
  hipDeviceSynchronize();
  hipMemcpy(&host_value,device_value,sizeof(int32_t),hipMemcpyDeviceToHost);
  hipFree(device_value);
  return host_value;
}

int32_t int32::le(const int16 &other) const {
  int32_t host_value;
  int32_t *device_value;
  hipMalloc(&device_value,sizeof(int32_t));
  leKernel<<<1,1>>>(this -> value,other.deviceValue(),device_value);
  hipGetLastError();
  hipDeviceSynchronize();
  hipMemcpy(&host_value,device_value,sizeof(int32_t),hipMemcpyDeviceToHost);
  hipFree(device_value);
  return host_value;
}

int64 int32::add(const int64 &other) const {
  int64 result(0);
  addKernel<<<1,1>>>(this -> value,other.deviceValue(),result.deviceValue());
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int64 int32::sub(const int64 &other) const {
  int64 result(0);
  subKernel<<<1,1>>>(this -> value,other.deviceValue(),result.deviceValue());
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int64 int32::mul(const int64 &other) const {
  int64 result(0);
  mulKernel<<<1,1>>>(this -> value,other.deviceValue(),result.deviceValue());
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int64 int32::tdiv(const int64 &other) const {
  int64 result(0);
  tdivKernel<<<1,1>>>(this -> value,other.deviceValue(),result.deviceValue());
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int64 int32::fdiv(const int64 &other) const {
  int64 result(0);
  fdivKernel<<<1,1>>>(this -> value,other.deviceValue(),result.deviceValue());
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int64 int32::mod(const int64 &other) const {
  int64 result(0);
  modKernel<<<1,1>>>(this -> value,other.deviceValue(),result.deviceValue());
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int64 int32::pow(const int64 &other) const {
  int64 result(0);
  powKernel<<<1,1>>>(this -> value,other.deviceValue(),result.deviceValue());
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int64_t int32::eq(const int64 &other) const {
  int64_t host_value;
  int64_t *device_value;
  hipMalloc(&device_value,sizeof(int64_t));
  eqKernel<<<1,1>>>(this -> value,other.deviceValue(),device_value);
  hipGetLastError();
  hipDeviceSynchronize();
  hipMemcpy(&host_value,device_value,sizeof(int64_t),hipMemcpyDeviceToHost);
  hipFree(device_value);
  return host_value;
}

int64_t int32::ne(const int64 &other) const {
  int64_t host_value;
  int64_t *device_value;
  hipMalloc(&device_value,sizeof(int64_t));
  neKernel<<<1,1>>>(this -> value,other.deviceValue(),device_value);
  hipGetLastError();
  hipDeviceSynchronize();
  hipMemcpy(&host_value,device_value,sizeof(int64_t),hipMemcpyDeviceToHost);
  hipFree(device_value);
  return host_value;
}

int64_t int32::gt(const int64 &other) const {
  int64_t host_value;
  int64_t *device_value;
  hipMalloc(&device_value,sizeof(int64_t));
  gtKernel<<<1,1>>>(this -> value,other.deviceValue(),device_value);
  hipGetLastError();
  hipDeviceSynchronize();
  hipMemcpy(&host_value,device_value,sizeof(int64_t),hipMemcpyDeviceToHost);
  hipFree(device_value);
  return host_value;
}

int64_t int32::ge(const int64 &other) const {
  int64_t host_value;
  int64_t *device_value;
  hipMalloc(&device_value,sizeof(int64_t));
  geKernel<<<1,1>>>(this -> value,other.deviceValue(),device_value);
  hipGetLastError();
  hipDeviceSynchronize();
  hipMemcpy(&host_value,device_value,sizeof(int64_t),hipMemcpyDeviceToHost);
  hipFree(device_value);
  return host_value;
}

int64_t int32::lt(const int64 &other) const {
  int64_t host_value;
  int64_t *device_value;
  hipMalloc(&device_value,sizeof(int64_t));
  ltKernel<<<1,1>>>(this -> value,other.deviceValue(),device_value);
  hipGetLastError();
  hipDeviceSynchronize();
  hipMemcpy(&host_value,device_value,sizeof(int64_t),hipMemcpyDeviceToHost);
  hipFree(device_value);
  return host_value;
}

int64_t int32::le(const int64 &other) const {
  int64_t host_value;
  int64_t *device_value;
  hipMalloc(&device_value,sizeof(int64_t));
  leKernel<<<1,1>>>(this -> value,other.deviceValue(),device_value);
  hipGetLastError();
  hipDeviceSynchronize();
  hipMemcpy(&host_value,device_value,sizeof(int64_t),hipMemcpyDeviceToHost);
  hipFree(device_value);
  return host_value;
}

int64 int64::add(const int64 &other) const {
  int64 result(0);
  addKernel<<<1,1>>>(this -> value,other.deviceValue(),result.value);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int64 int64::sub(const int64 &other) const {
  int64 result(0);
  subKernel<<<1,1>>>(this -> value,other.deviceValue(),result.value);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int64 int64::mul(const int64 &other) const {
  int64 result(0);
  mulKernel<<<1,1>>>(this -> value,other.deviceValue(),result.value);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int64 int64::tdiv(const int64 &other) const {
  int64 result(0);
  tdivKernel<<<1,1>>>(this -> value,other.value,result.value);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int64 int64::fdiv(const int64 &other) const {
  int64 result(0);
  fdivKernel<<<1,1>>>(this -> value,other.value,result.value);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int64 int64::mod(const int64 &other) const {
  int64 result(0);
  modKernel<<<1,1>>>(this -> value,other.value,result.value);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int64 int64::pow(const int64 &other) const {
  int64 result(0);
  powKernel<<<1,1>>>(this -> value,other.value,result.value);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int64_t int64::eq(const int64 &other) const {
  int64_t host_value;
  int64_t *device_value;
  hipMalloc(&device_value,sizeof(int64_t));
  eqKernel<<<1,1>>>(this -> value,other.value,device_value);
  hipGetLastError();
  hipDeviceSynchronize();
  hipMemcpy(&host_value,device_value,sizeof(int64_t),hipMemcpyDeviceToHost);
  hipFree(device_value);
  return host_value;
}

int64_t int64::ne(const int64 &other) const {
  int64_t host_value;
  int64_t *device_value;
  hipMalloc(&device_value,sizeof(int64_t));
  neKernel<<<1,1>>>(this -> value,other.value,device_value);
  hipGetLastError();
  hipDeviceSynchronize();
  hipMemcpy(&host_value,device_value,sizeof(int64_t),hipMemcpyDeviceToHost);
  hipFree(device_value);
  return host_value;
}

int64_t int64::gt(const int64 &other) const {
  int64_t host_value;
  int64_t *device_value;
  hipMalloc(&device_value,sizeof(int64_t));
  gtKernel<<<1,1>>>(this -> value,other.value,device_value);
  hipGetLastError();
  hipDeviceSynchronize();
  hipMemcpy(&host_value,device_value,sizeof(int64_t),hipMemcpyDeviceToHost);
  hipFree(device_value);
  return host_value;
}

int64_t int64::ge(const int64 &other) const {
  int64_t host_value;
  int64_t *device_value;
  hipMalloc(&device_value,sizeof(int64_t));
  geKernel<<<1,1>>>(this -> value,other.value,device_value);
  hipGetLastError();
  hipDeviceSynchronize();
  hipMemcpy(&host_value,device_value,sizeof(int64_t),hipMemcpyDeviceToHost);
  hipFree(device_value);
  return host_value;
}

int64_t int64::lt(const int64 &other) const {
  int64_t host_value;
  int64_t *device_value;
  hipMalloc(&device_value,sizeof(int64_t));
  ltKernel<<<1,1>>>(this -> value,other.value,device_value);
  hipGetLastError();
  hipDeviceSynchronize();
  hipMemcpy(&host_value,device_value,sizeof(int64_t),hipMemcpyDeviceToHost);
  hipFree(device_value);
  return host_value;
}

int64_t int64::le(const int64 &other) const {
  int64_t host_value;
  int64_t *device_value;
  hipMalloc(&device_value,sizeof(int64_t));
  leKernel<<<1,1>>>(this -> value,other.value,device_value);
  hipGetLastError();
  hipDeviceSynchronize();
  hipMemcpy(&host_value,device_value,sizeof(int64_t),hipMemcpyDeviceToHost);
  hipFree(device_value);
  return host_value;
}

int64 int64::abs() const {
  int64 result(0);
  absKernel<<<1,1>>>(this -> value,result.value);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int64 int64::neg() const {
  int64 result(0);
  negKernel<<<1,1>>>(this -> value,result.value);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int64 int64::pos() const {
  int64 result(0);
  posKernel<<<1,1>>>(this -> value,result.value);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int64 int64::add(const int32 &other) const {
  int64 result(0);
  addKernel<<<1,1>>>(this -> value,other.deviceValue(),result.value);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int64 int64::sub(const int32 &other) const {
  int64 result(0);
  subKernel<<<1,1>>>(this -> value,other.deviceValue(),result.value);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int64 int64::mul(const int32 &other) const {
  int64 result(0);
  mulKernel<<<1,1>>>(this -> value,other.deviceValue(),result.value);
  hipGetLastError();
  hipDeviceSynchronize();
  return resultl;
}

int64 int64::tdiv(const int32 &other) const {
  int64 result(0);
  tdivKernel<<<1,1>>>(this -> value,other.deviceValue(),result.value);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int64 int64::fdiv(const int32 &other) const {
  int64 result(0);
  fdivKernel<<<1,1>>>(this -> value,other.deviceValue(),result.value);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int64 int64::mod(const int32 &other) const {
  int64 result(0);
  modKernel<<<1,1>>>(this -> value,other.deviceValue(),result.value);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int64 int64::pow(const int32 &other) const {
  int64 result(0);
  powKernel<<<1,1>>>(this -> value,other.deviceValue(),result.value);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}
















#include "hip/hip_runtime.h"
#include <iostream>
#include "intn.cu"
using namespace std;

int main(){
  int16 x(2);
  int32 a(3);
  int64 u(2);

  int16_t v = 2;
  int64_t w = 12;

  cout << x.add(u).getValue() << endl;
  cout << x.sub(u).getValue() << endl;
  cout << v + w << endl;


  cout << a.eq(u) << endl;
//   cout << x.getValue() << " + " << a.getValue() << " = " << x.add(a).getValue() << endl;
//   cout << x.getValue() << " - " << a.getValue() << " = " << x.sub(a).getValue() << endl;
//   cout << x.getValue() << " * " << a.getValue() << " = " << x.mul(a).getValue() << endl;
//   cout << x.getValue() << " / " << a.getValue() << " = " << x.tdiv(a).getValue() << endl;
//   cout << x.getValue() << " // " << a.getValue() << " = " << x.fdiv(a).getValue() << endl;
//   cout << x.getValue() << " % " << a.getValue() << " = " << x.mod(a).getValue() << endl;
//   cout << x.getValue() << " ^ " << a.getValue() << " = " << x.pow(a).getValue() << endl;


//  cout << x.getValue() << " == " << a.getValue() << " = " << x.eq(a) << endl;
//  cout << x.getValue() << " != " << a.getValue() << " = " << x.ne(a) << endl;
//  cout << x.getValue() << " > " << a.getValue() << " = " << x.gt(a) << endl;
//  cout << x.getValue() << " >= " << a.getValue() << " = " << x.ge(a) << endl;
//  cout << x.getValue() << " < " << a.getValue() << " = " << x.lt(a) << endl;
//  cout << x.getValue() << " <= " << a.getValue() << " = " << x.le(a) << endl;

  // cout << x.sub(u).getValue() << endl;
  // cout << x.mul(u).getValue() << endl;
  // cout << x.tdiv(u).getValue() << endl;

  // int16_t x = 2;
  // int64_t y = 12;
  // int16_t z = 13;

  // cout << x+y << endl;
  // cout << x+z << endl;

  return 0;
}

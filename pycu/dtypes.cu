#include <hip/hip_runtime.h>


// CUDA kernels starts from here


__global__ void addKernel(const int16_t *x,const int16_t *y,int16_t *z){ *z = *x + *y; }
__global__ void subKernel(const int16_t *x,const int16_t *y,int16_t *z){ *z = *x - *y; }
__global__ void mulKernel(const int16_t *x,const int16_t *y,int16_t *z){ *z = *x * *y; }
__global__ void tdivKernel(const int16_t *x,const int16_t *y,int16_t *z){ *z = *x / *y; }
__global__ void fdivKernel(const int16_t *x,const int16_t *y,int16_t *z){ *z = *x / *y; }
__global__ void modKernel(const int16_t *x,const int16_t *y,int16_t *z){ *z = *x % *y; }
__global__ void powKernel(const int16_t *x,const int16_t *y,int16_t *z){ *z = pow(*x,*y); }
__global__ void absKernel(const int16_t *x,int16_t *z){
  if(*x < 0){ *z = -*x; }
  else{ *z = *x; }
}
__global__ void negKernel(const int16_t *x,int16_t *z){ *z = -*x; }
__global__ void posKernel(const int16_t *x,int16_t *z){ *z = +*x; }
__global__ void eqKernel(const int16_t *x,const int16_t *y,int16_t *z){ *z = (*x == *y) ? 1 : 0; }
__global__ void neKernel(const int16_t *x,const int16_t *y,int16_t *z){ *z = (*x != *y) ? 1 : 0; }
__global__ void gtKernel(const int16_t *x,const int16_t *y,int16_t *z){ *z = (*x > *y) ? 1 : 0; }
__global__ void geKernel(const int16_t *x,const int16_t *y,int16_t *z){ *z = (*x >= *y) ? 1 : 0; }
__global__ void ltKernel(const int16_t *x,const int16_t *y,int16_t *z){ *z = (*x < *y) ? 1 : 0; }
__global__ void leKernel(const int16_t *x,const int16_t *y,int16_t *z){ *z = (*x <= *y) ? 1 : 0; }

__global__ void addKernel(const int32_t *x,const int32_t *y,int32_t *z){ *z = *x + *y; }
__global__ void subKernel(const int32_t *x,const int32_t *y,int32_t *z){ *z = *x - *y; }
__global__ void mulKernel(const int32_t *x,const int32_t *y,int32_t *z){ *z = *x * *y; }
__global__ void tdivKernel(const int32_t *x,const int32_t *y,int32_t *z){ *z = *x / *y; }
__global__ void fdivKernel(const int32_t *x,const int32_t *y,int32_t *z){ *z = *x / *y; }
__global__ void modKernel(const int32_t *x,const int32_t *y,int32_t *z){ *z = *x % *y; }
__global__ void powKernel(const int32_t *x,const int32_t *y,int32_t *z){ *z = pow(*x,*y); }
__global__ void absKernel(const int32_t *x,int32_t *z){
  if(*x < 0){ *z = -*x; }
  else{ *z = *x; }
}
__global__ void negKernel(const int32_t *x,int32_t *z){ *z = -*x; }
__global__ void posKernel(const int32_t *x,int32_t *z){ *z = +*x; }
__global__ void eqKernel(const int32_t *x,const int32_t *y,int32_t *z){ *z = (*x == *y) ? 1 : 0; }
__global__ void neKernel(const int32_t *x,const int32_t *y,int32_t *z){ *z = (*x != *y) ? 1 : 0; }
__global__ void gtKernel(const int32_t *x,const int32_t *y,int32_t *z){ *z = (*x > *y) ? 1 : 0; }
__global__ void geKernel(const int32_t *x,const int32_t *y,int32_t *z){ *z = (*x >= *y) ? 1 : 0; }
__global__ void ltKernel(const int32_t *x,const int32_t *y,int32_t *z){ *z = (*x < *y) ? 1 : 0; }
__global__ void leKernel(const int32_t *x,const int32_t *y,int32_t *z){ *z = (*x <= *y) ? 1 : 0; }

__global__ void addKernel(const int64_t *x,const int64_t *y,int64_t *z){ *z = *x + *y; }
__global__ void subKernel(const int64_t *x,const int64_t *y,int64_t *z){ *z = *x - *y; }
__global__ void mulKernel(const int64_t *x,const int64_t *y,int64_t *z){ *z = *x * *y; }
__global__ void tdivKernel(const int64_t *x,const int64_t *y,int64_t *z){ *z = *x / *y; }
__global__ void fdivKernel(const int64_t *x,const int64_t *y,int64_t *z){ *z = *x / *y; }
__global__ void modKernel(const int64_t *x,const int64_t *y,int64_t *z){ *z = *x % *y; }
__global__ void powKernel(const int64_t *x,const int64_t *y,int64_t *z){ *z = pow(*x,*y); }
__global__ void absKernel(const int64_t *x,int64_t *z){
  if(*x < 0){ *z = -*x; }
  else{ *z = *x; }
}
__global__ void negKernel(const int64_t *x,int64_t *z){ *z = -*x; }
__global__ void posKernel(const int64_t *x,int64_t *z){ *z = +*x; }
__global__ void eqKernel(const int64_t *x,const int64_t *y,int64_t *z){ *z = (*x == *y) ? 1 : 0; }
__global__ void neKernel(const int64_t *x,const int64_t *y,int64_t *z){ *z = (*x != *y) ? 1 : 0; }
__global__ void gtKernel(const int64_t *x,const int64_t *y,int64_t *z){ *z = (*x > *y) ? 1 : 0; }
__global__ void geKernel(const int64_t *x,const int64_t *y,int64_t *z){ *z = (*x >= *y) ? 1 : 0; }
__global__ void ltKernel(const int64_t *x,const int64_t *y,int64_t *z){ *z = (*x < *y) ? 1 : 0; }
__global__ void leKernel(const int64_t *x,const int64_t *y,int64_t *z){ *z = (*x <= *y) ? 1 : 0; }


// CUDA kernel ends here


// predefined classes
class int16 ;
class int32 ;
class int64 ;




class int16{
  private:
    int16_t *val;
  
  public:
    int16(const int16_t &value){
      hipMalloc(&val,sizeof(int16_t));
      hipMemcpy(val,&value,sizeof(int16_t),hipMemcpyHostToDevice);
    }

    ~int16(){ if(val){ hipFree(val); } }

    int16_t getValue() const {
      int16_t host_value;
      hipMemcpy(&host_value,val,sizeof(int16_t),hipMemcpyDeviceToHost);
      return host_value;
    }

    int16_t *deviceValue() const { return this -> val; }
/*
    int32 to_int32() const {
      int16_t buff = getValue();
      return int32(static_cast<int32_t>(buff));
    }
*/
    int16 add(const int16 &other) const ;
    int16 sub(const int16 &other) const ;
    int16 mul(const int16 &other) const ;
    int16 tdiv(const int16 &other) const ;
    int16 fdiv(const int16 &other) const ;
    int16 mod(const int16 &other) const ;
    int16 pow(const int16 &other) const ;

    int16 abs() const ;
    int16 neg() const ;
    int16 pos() const ;

    int16_t eq(const int16 &other) const ;
    int16_t ne(const int16 &other) const ;
    int16_t gt(const int16 &other) const ;
    int16_t ge(const int16 &other) const ;
    int16_t lt(const int16 &other) const ;
    int16_t le(const int16 &other) const ;
};


class int32{
  private:
   int32_t *val;

  public:
    int32(const int32_t &value){
      hipMalloc(&val,sizeof(int32_t));
      hipMemcpy(val,&value,sizeof(int32_t),hipMemcpyHostToDevice);
    }

    ~int32(){ if(val){ hipFree(val); } }

    int32_t getValue() const {
      int32_t host_value;
      hipMemcpy(&host_value,val,sizeof(int32_t),hipMemcpyDeviceToHost);
      return host_value;
    }

    int32_t *deviceValue() const { return this -> val; }

    int32 add(const int32 &other) const ;
    int32 sub(const int32 &other) const ;
    int32 mul(const int32 &other) const ;
    int32 tdiv(const int32 &other) const ;
    int32 fdiv(const int32 &other) const ;
    int32 mod(const int32 &other) const ;
    int32 pow(const int32 &other) const ;

    int32 abs() const ;
    int32 neg() const ;
    int32 pos() const ;

    int32_t eq(const int32 &other) const ;
    int32_t ne(const int32 &other) const ;
    int32_t gt(const int32 &other) const ;
    int32_t ge(const int32 &other) const ;
    int32_t lt(const int32 &other) const ;
    int32_t le(const int32 &other) const ;
    
};


class int64{
  private:
    int64_t *val;

  public:
    int64(const int64_t &value){
      hipMalloc(&val,sizeof(int64_t));
      hipMemcpy(val,&value,sizeof(int64_t),hipMemcpyHostToDevice);
    }

    ~int64(){ if(val){ hipFree(val); } }

    int64_t getValue() const {
      int64_t host_value;
      hipMemcpy(&host_value,val,sizeof(int64_t),hipMemcpyDeviceToHost);
      return host_value;
    }

    int64_t *deviceValue() const { return this -> val; }

    int64 add(const int64 &other) const ;
    int64 sub(const int64 &other) const ;
    int64 mul(const int64 &other) const ;
    int64 tdiv(const int64 &other) const ;
    int64 fdiv(const int64 &other) const ;
    int64 mod(const int64 &other) const ;
    int64 pow(const int64 &other) const ;

    int64 abs() const ;
    int64 neg() const ;
    int64 pos() const ;

    int64_t eq(const int64 &other) const ;
    int64_t ne(const int64 &other) const ;
    int64_t gt(const int64 &other) const ;
    int64_t ge(const int64 &other) const ;
    int64_t lt(const int64 &other) const ;
    int64_t le(const int64 &other) const ;
};


int16 int16::add(const int16 &other) const {
  int16 result(0);
  addKernel<<<1,1>>>(this -> val,other.val,result.val);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int16 int16::sub(const int16 &other) const {
  int16 result(0);
  subKernel<<<1,1>>>(this -> val,other.val,result.val);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int16 int16::mul(const int16 &other) const {
  int16 result(0);
  mulKernel<<<1,1>>>(this -> val,other.val,result.val);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int16 int16::tdiv(const int16 &other) const {
  int16 result(0);
  tdivKernel<<<1,1>>>(this -> val,other.val,result.val);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int16 int16::fdiv(const int16 &other) const {
  int16 result(0);
  fdivKernel<<<1,1>>>(this -> val,other.val,result.val);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int16 int16::mod(const int16 &other) const {
  int16 result(0);
  modKernel<<<1,1>>>(this -> val,other.val,result.val);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int16 int16::pow(const int16 &other) const {
  int16 result(0);
  powKernel<<<1,1>>>(this -> val,other.val,result.val);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int16 int16::abs() const {
  int16 result(0);
  absKernel<<<1,1>>>(this -> val,result.val);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int16 int16::neg() const {
  int16 result(0);
  negKernel<<<1,1>>>(this -> val,result.val);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int16 int16::pos() const {
  int16 result(0);
  posKernel<<<1,1>>>(this -> val,result.val);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int16_t int16::eq(const int16 &other) const {
  int16_t host_value;
  int16_t *device_value;
  hipMalloc(&device_value,sizeof(int16_t));
  eqKernel<<<1,1>>>(this -> val,other.val,device_value);
  hipGetLastError();
  hipDeviceSynchronize();
  hipMemcpy(&host_value,device_value,sizeof(int16_t),hipMemcpyDeviceToHost);
  hipFree(device_value);
  return host_value;
}

int16_t int16::ne(const int16 &other) const {
  int16_t host_value;
  int16_t *device_value;
  hipMalloc(&device_value,sizeof(int16_t));
  neKernel<<<1,1>>>(this -> val,other.val,device_value);
  hipGetLastError();
  hipDeviceSynchronize();
  hipMemcpy(&host_value,device_value,sizeof(int16_t),hipMemcpyDeviceToHost);
  hipFree(device_value);
  return host_value;
}

int16_t int16::gt(const int16 &other) const {
  int16_t host_value;
  int16_t *device_value;
  hipMalloc(&device_value,sizeof(int16_t));
  gtKernel<<<1,1>>>(this -> val,other.val,device_value);
  hipGetLastError();
  hipDeviceSynchronize();
  hipMemcpy(&host_value,device_value,sizeof(int16_t),hipMemcpyDeviceToHost);
  hipFree(device_value);
  return host_value;
}

int16_t int16::ge(const int16 &other) const {
  int16_t host_value;
  int16_t *device_value;
  hipMalloc(&device_value,sizeof(int16_t));
  geKernel<<<1,1>>>(this -> val,other.val,device_value);
  hipGetLastError();
  hipDeviceSynchronize();
  hipMemcpy(&host_value,device_value,sizeof(int16_t),hipMemcpyDeviceToHost);
  hipFree(device_value);
  return host_value;
}

int16_t int16::lt(const int16 &other) const {
  int16_t host_value;
  int16_t *device_value;
  hipMalloc(&device_value,sizeof(int16_t));
  ltKernel<<<1,1>>>(this -> val,other.val,device_value);
  hipGetLastError();
  hipDeviceSynchronize();
  hipMemcpy(&host_value,device_value,sizeof(int16_t),hipMemcpyDeviceToHost);
  hipFree(device_value);
  return host_value;
}

int16_t int16::le(const int16 &other) const {
  int16_t host_value;
  int16_t *device_value;
  hipMalloc(&device_value,sizeof(int16_t));
  leKernel<<<1,1>>>(this -> val,other.val,device_value);
  hipGetLastError();
  hipDeviceSynchronize();
  hipMemcpy(&host_value,device_value,sizeof(int16_t),hipMemcpyDeviceToHost);
  hipFree(device_value);
  return host_value;
}




int32 int32::add(const int32 &other) const {
  int32 result(0);
  addKernel<<<1,1>>>(this -> val,other.val,result.val);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int32 int32::sub(const int32 &other) const {
  int32 result(0);
  subKernel<<<1,1>>>(this -> val,other.val,result.val);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int32 int32::mul(const int32 &other) const {
  int32 result(0);
  mulKernel<<<1,1>>>(this -> val,other.val,result.val);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int32 int32::tdiv(const int32 &other) const {
  int32 result(0);
  tdivKernel<<<1,1>>>(this -> val,other.val,result.val);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int32 int32::fdiv(const int32 &other) const {
  int32 result(0);
  fdivKernel<<<1,1>>>(this -> val,other.val,result.val);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int32 int32::mod(const int32 &other) const {
  int32 result(0);
  modKernel<<<1,1>>>(this -> val,other.val,result.val);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int32 int32::pow(const int32 &other) const {
  int32 result(0);
  powKernel<<<1,1>>>(this -> val,other.val,result.val);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int32 int32::abs() const {
  int32 result(0);
  absKernel<<<1,1>>>(this -> val,result.val);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int32 int32::neg() const {
  int32 result(0);
  negKernel<<<1,1>>>(this -> val,result.val);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int32 int32::pos() const {
  int32 result(0);
  posKernel<<<1,1>>>(this -> val,result.val);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int32_t int32::eq(const int32 &other) const {
  int32_t host_value;
  int32_t *device_value;
  hipMalloc(&device_value,sizeof(int32_t));
  eqKernel<<<1,1>>>(this -> val,other.val,device_value);
  hipGetLastError();
  hipDeviceSynchronize();
  hipMemcpy(&host_value,device_value,sizeof(int32_t),hipMemcpyDeviceToHost);
  hipFree(device_value);
  return host_value;
}

int32_t int32::ne(const int32 &other) const {
  int32_t host_value;
  int32_t *device_value;
  hipMalloc(&device_value,sizeof(int32_t));
  neKernel<<<1,1>>>(this -> val,other.val,device_value);
  hipGetLastError();
  hipDeviceSynchronize();
  hipMemcpy(&host_value,device_value,sizeof(int32_t),hipMemcpyDeviceToHost);
  hipFree(device_value);
  return host_value;
}

int32_t int32::gt(const int32 &other) const {
  int32_t host_value;
  int32_t *device_value;
  hipMalloc(&device_value,sizeof(int32_t));
  gtKernel<<<1,1>>>(this -> val,other.val,device_value);
  hipGetLastError();
  hipDeviceSynchronize();
  hipMemcpy(&host_value,device_value,sizeof(int32_t),hipMemcpyDeviceToHost);
  hipFree(device_value);
  return host_value;
}

int32_t int32::ge(const int32 &other) const {
  int32_t host_value;
  int32_t *device_value;
  hipMalloc(&device_value,sizeof(int32_t));
  geKernel<<<1,1>>>(this -> val,other.val,device_value);
  hipGetLastError();
  hipDeviceSynchronize();
  hipMemcpy(&host_value,device_value,sizeof(int32_t),hipMemcpyDeviceToHost);
  hipFree(device_value);
  return host_value;
}

int32_t int32::lt(const int32 &other) const {
  int32_t host_value;
  int32_t *device_value;
  hipMalloc(&device_value,sizeof(int32_t));
  ltKernel<<<1,1>>>(this -> val,other.val,device_value);
  hipGetLastError();
  hipDeviceSynchronize();
  hipMemcpy(&host_value,device_value,sizeof(int32_t),hipMemcpyDeviceToHost);
  hipFree(device_value);
  return host_value;
}

int32_t int32::le(const int32 &other) const {
  int32_t host_value;
  int32_t *device_value;
  hipMalloc(&device_value,sizeof(int32_t));
  leKernel<<<1,1>>>(this -> val,other.val,device_value);
  hipGetLastError();
  hipDeviceSynchronize();
  hipMemcpy(&host_value,device_value,sizeof(int32_t),hipMemcpyDeviceToHost);
  hipFree(device_value);
  return host_value;
}





int64 int64::add(const int64 &other) const {
  int64 result(0);
  addKernel<<<1,1>>>(this -> val,other.val,result.val);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int64 int64::sub(const int64 &other) const {
  int64 result(0);
  subKernel<<<1,1>>>(this -> val,other.val,result.val);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int64 int64::mul(const int64 &other) const {
  int64 result(0);
  mulKernel<<<1,1>>>(this -> val,other.val,result.val);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int64 int64::tdiv(const int64 &other) const {
  int64 result(0);
  tdivKernel<<<1,1>>>(this -> val,other.val,result.val);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int64 int64::fdiv(const int64 &other) const {
  int64 result(0);
  fdivKernel<<<1,1>>>(this -> val,other.val,result.val);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int64 int64::mod(const int64 &other) const {
  int64 result(0);
  modKernel<<<1,1>>>(this -> val,other.val,result.val);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int64 int64::pow(const int64 &other) const {
  int64 result(0);
  powKernel<<<1,1>>>(this -> val,other.val,result.val);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int64 int64::abs() const {
  int64 result(0);
  absKernel<<<1,1>>>(this -> val,result.val);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int64 int64::neg() const {
  int64 result(0);
  negKernel<<<1,1>>>(this -> val,result.val);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int64 int64::pos() const {
  int64 result(0);
  posKernel<<<1,1>>>(this -> val,result.val);
  hipGetLastError();
  hipDeviceSynchronize();
  return result;
}

int64_t int64::eq(const int64 &other) const {
  int64_t host_value;
  int64_t *device_value;
  hipMalloc(&device_value,sizeof(int64_t));
  eqKernel<<<1,1>>>(this -> val,other.val,device_value);
  hipGetLastError();
  hipDeviceSynchronize();
  hipMemcpy(&host_value,device_value,sizeof(int64_t),hipMemcpyDeviceToHost);
  hipFree(device_value);
  return host_value;
}

int64_t int64::ne(const int64 &other) const {
  int64_t host_value;
  int64_t *device_value;
  hipMalloc(&device_value,sizeof(int64_t));
  neKernel<<<1,1>>>(this -> val,other.val,device_value);
  hipGetLastError();
  hipDeviceSynchronize();
  hipMemcpy(&host_value,device_value,sizeof(int64_t),hipMemcpyDeviceToHost);
  hipFree(device_value);
  return host_value;
}

int64_t int64::gt(const int64 &other) const {
  int64_t host_value;
  int64_t *device_value;
  hipMalloc(&device_value,sizeof(int64_t));
  gtKernel<<<1,1>>>(this -> val,other.val,device_value);
  hipGetLastError();
  hipDeviceSynchronize();
  hipMemcpy(&host_value,device_value,sizeof(int64_t),hipMemcpyDeviceToHost);
  hipFree(device_value);
  return host_value;
}

int64_t int64::ge(const int64 &other) const {
  int64_t host_value;
  int64_t *device_value;
  hipMalloc(&device_value,sizeof(int64_t));
  geKernel<<<1,1>>>(this -> val,other.val,device_value);
  hipGetLastError();
  hipDeviceSynchronize();
  hipMemcpy(&host_value,device_value,sizeof(int64_t),hipMemcpyDeviceToHost);
  hipFree(device_value);
  return host_value;
}

int64_t int64::lt(const int64 &other) const {
  int64_t host_value;
  int64_t *device_value;
  hipMalloc(&device_value,sizeof(int64_t));
  ltKernel<<<1,1>>>(this -> val,other.val,device_value);
  hipGetLastError();
  hipDeviceSynchronize();
  hipMemcpy(&host_value,device_value,sizeof(int64_t),hipMemcpyDeviceToHost);
  hipFree(device_value);
  return host_value;
}

int64_t int64::le(const int64 &other) const {
  int64_t host_value;
  int64_t *device_value;
  hipMalloc(&device_value,sizeof(int64_t));
  leKernel<<<1,1>>>(this -> val,other.val,device_value);
  hipGetLastError();
  hipDeviceSynchronize();
  hipMemcpy(&host_value,device_value,sizeof(int64_t),hipMemcpyDeviceToHost);
  hipFree(device_value);
  return host_value;
}



extern "C"{
  int16 *int16_new(int16_t value){ return new int16(value); }
  void int16_delete(int16 *value){ delete value; }
  int16_t int16_value(int16 *value){ return value -> getValue(); }

  int16 *int16_add(int16 *x,int16 *y){ return new int16(x -> add(*y)); }
  int16 *int16_sub(int16 *x,int16 *y){ return new int16(x -> sub(*y)); }
  int16 *int16_mul(int16 *x,int16 *y){ return new int16(x -> mul(*y)); }
  int16 *int16_tdiv(int16 *x,int16 *y){ return new int16(x -> tdiv(*y)); }
  int16 *int16_fdiv(int16 *x,int16 *y){ return new int16(x -> fdiv(*y)); }
  int16 *int16_mod(int16 *x,int16 *y){ return new int16(x -> mod(*y)); }
  int16 *int16_pow(int16 *x,int16 *y){ return new int16(x -> pow(*y)); }
  int16 *int16_abs(int16 *x){ return new int16(x -> abs()); }
  int16 *int16_neg(int16 *x){ return new int16(x -> neg()); }
  int16 *int16_pos(int16 *x){ return new int16(x -> pos()); }
  int16_t int16_eq(int16 *x,int16 *y){ return x -> eq(*y); }
  int16_t int16_ne(int16 *x,int16 *y){ return x -> ne(*y); }
  int16_t int16_gt(int16 *x,int16 *y){ return x -> gt(*y); }
  int16_t int16_ge(int16 *x,int16 *y){ return x -> ge(*y); }
  int16_t int16_lt(int16 *x,int16 *y){ return x -> lt(*y); }
  int16_t int16_le(int16 *x,int16 *y){ return x -> le(*y); }


  int32 *int32_new(int32_t value){ return new int32(value); }
  void int32_delete(int32 *value){ delete value; }
  int32_t int32_value(int32 *value){ return value -> getValue(); }

  int32 *int32_add(int32 *x,int32 *y){ return new int32(x -> add(*y)); }
  int32 *int32_sub(int32 *x,int32 *y){ return new int32(x -> sub(*y)); }
  int32 *int32_mul(int32 *x,int32 *y){ return new int32(x -> mul(*y)); }
  int32 *int32_tdiv(int32 *x,int32 *y){ return new int32(x -> tdiv(*y)); }
  int32 *int32_fdiv(int32 *x,int32 *y){ return new int32(x -> fdiv(*y)); }
  int32 *int32_mod(int32 *x,int32 *y){ return new int32(x -> mod(*y)); }
  int32 *int32_pow(int32 *x,int32 *y){ return new int32(x -> pow(*y)); }
  int32 *int32_abs(int32 *x){ return new int32(x -> abs()); }
  int32 *int32_neg(int32 *x){ return new int32(x -> neg()); }
  int32 *int32_pos(int32 *x){ return new int32(x -> pos()); }
  int32_t int32_eq(int32 *x,int32 *y){ return x -> eq(*y); }
  int32_t int32_ne(int32 *x,int32 *y){ return x -> ne(*y); }
  int32_t int32_gt(int32 *x,int32 *y){ return x -> gt(*y); }
  int32_t int32_ge(int32 *x,int32 *y){ return x -> ge(*y); }
  int32_t int32_lt(int32 *x,int32 *y){ return x -> lt(*y); }
  int32_t int32_le(int32 *x,int32 *y){ return x -> le(*y); }


  int64 *int64_new(int64_t value){ return new int64(value); }
  void int64_delete(int64 *value){ delete value; }
  int64_t int64_value(int64 *value){ return value -> getValue(); }

  int64 *int64_add(int64 *x,int64 *y){ return new int64(x -> add(*y)); }
  int64 *int64_sub(int64 *x,int64 *y){ return new int64(x -> sub(*y)); }
  int64 *int64_mul(int64 *x,int64 *y){ return new int64(x -> mul(*y)); }
  int64 *int64_tdiv(int64 *x,int64 *y){ return new int64(x -> tdiv(*y)); }
  int64 *int64_fdiv(int64 *x,int64 *y){ return new int64(x -> fdiv(*y)); }
  int64 *int64_mod(int64 *x,int64 *y){ return new int64(x -> mod(*y)); }
  int64 *int64_pow(int64 *x,int64 *y){ return new int64(x -> pow(*y)); }
  int64 *int64_abs(int64 *x){ return new int64(x -> abs()); }
  int64 *int64_neg(int64 *x){ return new int64(x -> neg()); }
  int64 *int64_pos(int64 *x){ return new int64(x -> pos()); }
  int64_t int64_eq(int64 *x,int64 *y){ return x -> eq(*y); }
  int64_t int64_ne(int64 *x,int64 *y){ return x -> ne(*y); }
  int64_t int64_gt(int64 *x,int64 *y){ return x -> gt(*y); }
  int64_t int64_ge(int64 *x,int64 *y){ return x -> ge(*y); }
  int64_t int64_lt(int64 *x,int64 *y){ return x -> lt(*y); }
  int64_t int64_le(int64 *x,int64 *y){ return x -> le(*y); }
}

